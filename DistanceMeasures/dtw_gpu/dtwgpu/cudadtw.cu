
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern __shared__ float array[];

__global__ void calc_dtw (unsigned SRC_LEN,unsigned TRG_LEN,unsigned TRG_COT,
    float *S, float *TT, float *Result)
{
    // blockDim.x = TRG_LEN*TRG_COT
    float* path_h1 = (float*)array;
    float* path_h2 = (float*)&path_h1[blockDim.x];
    float* dist    = (float*)&path_h2[blockDim.x];

    //int blockId = (blockIdx.y*gridDim.x + blockIdx.x);
    //int G_idx   = blockId *blockDim.x +threadIdx.x;
    float *T =     TT + (blockIdx.y*gridDim.x +blockIdx.x) *blockDim.x;
    float *R = Result + (blockIdx.y*gridDim.x +blockIdx.x) *TRG_COT;
    
    float *ex;
    int i,j;
    
    int sub_x = threadIdx.x % TRG_LEN;
    int x_cot = threadIdx.x / TRG_LEN;
    
    // first line speical, do first
    // 1. paralle, first line's every element's dist
    dist[threadIdx.x] = (S[0] -T[threadIdx.x])
                       *(S[0] -T[threadIdx.x]);
    __syncthreads();
    // 2. serie, first line's every element's serie's dist
    if (sub_x == 0){
        path_h1[threadIdx.x] = dist[threadIdx.x];
        for (i=1; i <TRG_LEN; i++) {
            path_h1[i +threadIdx.x] = path_h1[i-1 +threadIdx.x]
                                    +    dist[i   +threadIdx.x];
        }
    }
    __syncthreads();
    
    for (i=1; i <SRC_LEN; i++){ // do circle
        // 1. paralle, calc itself's DISTANCE, for speed follow progress
        //    use memeory to rise speed
        dist[threadIdx.x] = (S[i] -T[threadIdx.x])
                           *(S[i] -T[threadIdx.x]);
        __syncthreads();
        // 2. paralle, get from upper line's "up","left-up"- the min dist
        if (sub_x == 0) // FIRST element speical, add DISTANCE here
            path_h2[threadIdx.x] =  path_h1[threadIdx.x] + dist[threadIdx.x];
        else
            path_h2[threadIdx.x] = min (path_h1[threadIdx.x],
                                        path_h1[threadIdx.x-1]);
        __syncthreads();
        // 2. serie, compare to left(front) element with myself, get the less value
        if (sub_x == 0) // the first element had plused dist, so can use here
            for (j=1; j<TRG_LEN; j++) 
                path_h2[j +threadIdx.x] = min(path_h2[j   +threadIdx.x], 
                                              path_h2[j-1 +threadIdx.x])
                                        + dist[j +threadIdx.x];
        __syncthreads();
        ex      = path_h2;
        path_h2 = path_h1;
        path_h1 = ex;
        __syncthreads();
    }
    // when all done, can return the result
    if (sub_x == 0){
        R[x_cot] = sqrt(ex[threadIdx.x +TRG_LEN -1]);
    }
}    
